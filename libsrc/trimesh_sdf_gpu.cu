#include "hip/hip_runtime.h"
#include "TriMesh.h"
#include "TriMesh_algo.h"
#include "kdTree_face.h"
#include "octree.h"
#include <numeric>
#include "hip/hip_runtime.h"
#include<hip/hip_runtime.h>
#include<>
#include<conio.h>
//#include "Vec.h"
using namespace std;
namespace trimesh {
	//#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
	//
	//	inline void gpuAssert(hipError_t code, char *file, int line, bool abort = true)
	//	{
	//		if (code != hipSuccess)
	//		{
	//			fprintf(stderr, "GPUassert: %s %s %dn\n", hipGetErrorString(code), file, line);
	//			//if (abort) { getch(); exit(code); }
	//		}
	//	}



	const int CONE_ANGLE = 60;
	const int RAYS_PER_ANGLE = 20;
	const int ANGLE_INTERVALS = 4;
	void vec_to_pointer(std::vector<vec>input, float***result) {
		int Ncol = 3;
		int Nrow = input.size();
		int nf = Nrow;
		*result = new float*[Nrow];
		for (int i = 0; i < Nrow; i++) {
			(*result)[i] = new float[Ncol];
			for (int j = 0; j < 3; j++) {
				(*result)[i][j] = input[i][j];
			}
		}
	};

	//rotation( vec dir, float angle, vec rotateAxis)
	//Goal: return the dirction vector after rotation *angle along *rotateaxis
	/*
	input:
	dir:		unit vector of direction
	angle:		Rotation angle in degree
	rotateAxis:	unit vector of the rotation axis
	*/
	inline vec rotation(vec dir, float angle, vec rotateAxis) {
		//vec oDir = dir - origin;
		float cos_angle = cos(angle*M_PIf / 180);
		float sin_angle = sin(angle*M_PIf / 180);
		//stack overflow 
		//https://stackoverflow.com/questions/42421611/3d-vector-rotation-in-c
		vec result = (cos_angle* dir) + ((rotateAxis CROSS dir)*sin_angle) + (rotateAxis * (rotateAxis ^ dir) *(1 - cos_angle));

		//wikipedia way
		vec Rr1 = vec(cos_angle + (rotateAxis[0] * rotateAxis[0])*(1 - cos_angle),
			(rotateAxis[0] * rotateAxis[1])*(1 - cos_angle) - rotateAxis[2] * sin_angle,
			(rotateAxis[0] * rotateAxis[2])*(1 - cos_angle) + rotateAxis[1] * sin_angle);

		vec Rr2 = vec((rotateAxis[0] * rotateAxis[1])*(1 - cos_angle) + rotateAxis[2] * sin_angle,
			cos_angle + (rotateAxis[1] * rotateAxis[1])*(1 - cos_angle),
			(rotateAxis[1] * rotateAxis[2])*(1 - cos_angle) - rotateAxis[0] * sin_angle);

		vec Rr3 = vec((rotateAxis[0] * rotateAxis[2])*(1 - cos_angle) - rotateAxis[1] * sin_angle,
			(rotateAxis[1] * rotateAxis[2])*(1 - cos_angle) + rotateAxis[0] * sin_angle,
			cos_angle + (rotateAxis[2] * rotateAxis[2])*(1 - cos_angle));

		result = vec(Rr1 ^ dir, Rr2 ^ dir, Rr3 ^ dir);
		//result = result + origin;
		normalize(result);
		return result;
	}
	//Goal: make a cone of rays
	/*input:
	halfAngle:	Half angle
	rings:		rays on
	v1,v2,v3:	points of the face
	*/
	inline void make_cone(float halfAngle, int rings, int intervals, vec origin, vec normal, vector<vec>& output) {
		output.resize(rings);

		//find perpendicular  vector as rotation axis
		vec rotateAxis = vec(1, 1, (-normal[0] - normal[1]) / normal[2]);
		normalize(rotateAxis);

		vec ray1 = rotation(normal, halfAngle, rotateAxis);

		for (int i = 0; i < rings; i++) {
			vec rayI = rotation(ray1, 360 * i / rings, normal);
			output[i] = rayI;
		}

	}
	int iDivUp(int hostPtr, int b) { return ((hostPtr % b) != 0) ? (hostPtr / b + 1) : (hostPtr / b); }

	__device__ void  cross_device(float* v1, float * v2, float(&result)[3]) {
		//float result[3];
		result[0] = v1[1] * v2[2] - v1[2] * v2[1];
		result[1] = v1[2] * v2[0] - v1[0] * v2[2];
		result[2] = v1[0] * v2[1] - v1[1] * v2[0];
		//return result;
	}

	__device__ float dot_device(float * v1, float *v2) {
		float sum = v1[0] * v2[0];
		for (size_t i = 1; i < 3; i++)
			sum += v1[i] * v2[i];
		return sum;
	}

	__device__ void plus_device(float *v1, float* v2, float(&result)[3], bool plus = true) {
		//float * result2 = new float[3];
		if (plus) {
			for (int i = 0; i < 3; i++) {
				result[i] = v1[i] + v2[i];
			}
		}
		else {
			for (int i = 0; i < 3; i++) {
				result[i] = v1[i] - v2[i];
			}
		}

	}

	__device__ void scale_device(float s, float* v1, float(&result)[3]) {
		//float * result = new float[3];
		for (int i = 0; i < 3; i++) {
			result[i] = s*v1[i];
		}
		//return result;
	}
	__device__ float len_device(float* v1) {
		float result = dot_device(v1, v1);
		return sqrt(result);
	}


	__device__ float triangle_inter_device(float * v1, float * v2, float* v3,
		float *dir, float * vertex) {
		float e[2][3];
		//e[0] = new float[3];
		//e[1] = new float[3];
		for (int i = 0; i < 3; i++) {
			e[0][i] = v2[i] - v1[i];
			e[1][i] = v3[i] - v1[i];
		}

		//float * pvec;
		//pvec = cross_device(e[1], dir);
		float  pvec[3];

		cross_device(e[1], dir, pvec);

		float det = dot_device(e[0], pvec);
		if (det < 1e-8 && det > -1e-8) {
			//std::cout << "parrel";
			return 0;
		}

		float inv_det = 1 / det;
		float tvec[3];
		plus_device(vertex, v1, tvec, false);
		float u = dot_device(tvec, pvec) * inv_det;
		if (u < 0 || u > 1) {
			return 0;
		}
		float qvec[3];
		cross_device(tvec, e[0], qvec);
		float v = dot_device(dir, qvec) * inv_det;
		if (v < 0 || u + v > 1) {
			return 0;
		}
		float t = inv_det * dot_device(e[1], qvec);

		//calculate the intersection points.
		//float * inter_point =plus( vertex , scale(t,dir));
		float result[3];
		scale_device(t, dir, result);
		return len_device(result);

	}

	__device__ float cal_dist_on_node_device(float * p, float * dir, float * pNormal,
		float * p1X, float * p1Y, float * p1Z,
		float * p2X, float * p2Y, float * p2Z,
		float * p3X, float * p3Y, float * p3Z,
		float * faceNX, float * faceNY, float * faceNZ, int size) {
		float temp = 10000000;
		for (int faceid = 0; faceid < size; faceid++) {
			float sum = faceNX[faceid] * pNormal[0] + faceNY[faceid] * pNormal[1] + faceNZ[faceid] * pNormal[2];
			float angle = acos(sum)* 180.0 / M_PIf;

			float v1[3] = { p1X[faceid] , p1Y[faceid] , p1Z[faceid] };
			float v2[3] = { p2X[faceid] , p2Y[faceid] , p2Z[faceid] };
			float v3[3] = { p3X[faceid] , p3Y[faceid] , p3Z[faceid] };
			//--Check whether the ray and the faces normal is above 90 degree.
			if (angle>90) {
				//int distance = 0;
				float distance = triangle_inter_device(v1, v2, v3, dir, p);
				if (distance != 0 && distance < temp) {
					temp = distance;
				}
			}
		}
		if (temp == 10000000)
			return -1;
		return temp;
	}

	__device__ int kd_right_device(int index) {
		return ((index + 1) * 2);
	}
	__device__ int kd_left_device(int index) {
		return ((index + 1) * 2) - 1;
	}

	__global__ void intersect_gpu_greater(float *pt1x, float *pt1y, float *pt1z,
		float *pt2x, float *pt2y, float *pt2z,
		float *pt3x, float *pt3y, float *pt3z,
		float *fNx, float *fNy, float *fNz,
		float *vx, float*vy, float*vz,
		float * normal_x, float *normal_y, float *normal_z,
		float *dirX, float *dirY, float *dirZ,
		float *minDist,
		int nv, int nRay,
		int nf_max, int nf_min)
	{
		//int i = threadIdx.x;

		//int i_face = blockIdx.x * blockDim.x + threadIdx.x;
		//int i_rays = blockIdx.z * blockDim.z + threadIdx.z;
		//int i_vertices = blockIdx.y * blockDim.y + threadIdx.y;

		int i_face = blockIdx.x * blockDim.x + threadIdx.x;
		int i_rays = blockIdx.z * blockDim.z + threadIdx.z;
		int i_vertices = (blockIdx.y * blockDim.y + threadIdx.y) + nf_min;

		if (i_face < nv && i_vertices<nf_max && i_rays<nRay) {
			int j = i_vertices*nRay + i_rays;
			float v1[3] = { pt1x[i_face] , pt1y[i_face] , pt1z[i_face] };
			float v2[3] = { pt2x[i_face] , pt2y[i_face] , pt2z[i_face] };
			float v3[3] = { pt3x[i_face] , pt3y[i_face] , pt3z[i_face] };


			float dir[3] = { dirX[j] , dirY[j] , dirZ[j] };
			float vertex[3] = { vx[i_vertices] , vy[i_vertices] , vz[i_vertices] };

			float sum = fNx[i_face] * normal_x[i_vertices] + fNy[i_face] * normal_y[i_vertices] + fNz[i_face] * normal_z[i_vertices];
			//float sum = fNx[1] * normal_x[1] + fNy[1] * normal_y[1] + fNz[1] * normal_z[1];
			float angle = acos(sum)* 180.0 / M_PIf;
			//float result;
			if (angle>90) {
				//int distance = 0;
				float distance = triangle_inter_device(v1, v2, v3, dir, vertex);
				if (distance != 0) {
					if (minDist[j] == 0) {
						minDist[j] = distance;
					}
					//else {
					//	if (distance < minDist[j]) {
					//		minDist[j] = distance;
					//	}
					//}
				}
				//else if (distance < minDist[j]) {
				//	minDist[j] = distance;
				//}
			}

		}

	}
	

	__global__ void intersect_gpu_kdtree(
		float *pt1x, float *pt1y, float *pt1z,
		float *pt2x, float *pt2y, float *pt2z,
		float *pt3x, float *pt3y, float *pt3z,
		float *fNx, float *fNy, float *fNz,
		float *split, int * split_axis, int* tri_count, int * tri_index,
		float *aabb_max, float *aabb_min,
		float *vx, float*vy, float*vz,
		float * normal_x, float *normal_y, float *normal_z,
		float *dirX, float *dirY, float *dirZ,
		float *minDist,
		int nv, int nRay,
		int n_kd)
	{

		int i_vertices = blockIdx.x * blockDim.x + threadIdx.x;
		int i_rays = (blockIdx.y * blockDim.y + threadIdx.y);

		if (i_vertices<nv && i_rays<nRay) {
			int j = i_vertices*nRay + i_rays;
			float dir[3] = { dirX[j] , dirY[j] , dirZ[j] };
			float vertex[3] = { vx[i_vertices] , vy[i_vertices] , vz[i_vertices] };
			float pNormal[3] = { normal_x[i_vertices],normal_y[i_vertices] , normal_z[i_vertices] };
			float t_i0[3];
			float t_i1[3];

			for (int i = 0; i < 3; i++) {
				if (vertex[i] == 0 || dir[i] == 0) {
					vertex[i] = 0; dir[i] = 0;
				}
				if (dir[i] >= 0) {
					t_i0[i] = (aabb_min[i] - vertex[i]) / dir[i];
					t_i1[i] = (aabb_max[i] - vertex[i]) / dir[i];
				}
				else {
					t_i0[i] = (aabb_max[i] - vertex[i]) / dir[i];
					t_i1[i] = (aabb_min[i] - vertex[i]) / dir[i];
				}
			}
			float t_min = max(t_i0[0], max(t_i0[1], t_i0[2]));
			float t_max = min(t_i1[0], min(t_i1[1], t_i1[2]));
			float global_max = t_max;
			int index = 0; float distance = -1;
			if (t_max > t_min) {
				int id_stack[64];
				int * stackPtr = id_stack; *stackPtr++ = -1;
				float t_min_stack[64]; float t_max_stack[64];
				float * stackPtr_tmin = t_min_stack; *stackPtr_tmin++ = -1.0;
				float * stackPtr_tmax = t_max_stack; *stackPtr_tmax++ = -1.0;
				while (distance <= 0 && index <= n_kd) {
					if (tri_count[index] == 0) {
						//--KD restart:
						if (t_max == global_max)
							break;
						else {
							t_min = t_max;
							t_max = global_max;
							index = 0;
						}
						//NULL node;
						index = *--stackPtr; t_min = *--stackPtr_tmin; t_max = *--stackPtr_tmax;
						if (index == -1)
							break;
					}
					else if (tri_count[index] != -1) {
						//is leaf 
						int leaf_size = tri_count[index];
						int leaf_index = tri_index[index];
						distance = cal_dist_on_node_device(vertex, dir, pNormal,
							&pt1x[leaf_index], &pt1y[leaf_index], &pt1z[leaf_index],
							&pt2x[leaf_index], &pt2y[leaf_index], &pt2z[leaf_index],
							&pt3x[leaf_index], &pt3y[leaf_index], &pt3z[leaf_index],
							&fNx[leaf_index], &fNy[leaf_index], &fNz[leaf_index],
							leaf_size);

						//change the index by POPPing the value in the stack
						//if (*(stackPtr - 1) != -1) {
						//--kd restart search:
						if (t_max == global_max)
							break;
						else {
							t_min = t_max;
							t_max = global_max;
							index = 0;
						}


						//index = *--stackPtr; t_min = *--stackPtr_tmin; t_max = *--stackPtr_tmax;
						////}
						//if (index == -1)
						//	break;

					}
					else {
						//NOT leaf
						int first_index, second_index;
						int axis = split_axis[index];
						float thit = (split[index] - vertex[axis]) / dir[axis];
						if ((split[index] - vertex[axis]) >= 0) {
							//first_index = kd_left_device(index); second_index = kd_right_device(index);
							first_index = ((index + 1) * 2) - 1; second_index = ((index + 1) * 2);
						}
						else {
							//first_index = kd_right_device(index); second_index = kd_left_device(index);
							first_index = ((index + 1) * 2); second_index = ((index + 1) * 2) - 1;
						}

						if (thit >= t_max || thit < 0)
							index = first_index;
						else if (thit <= t_min)
							index = second_index;
						else {
							//--push the index of Right node into Stack
							//*stackPtr++ = second_index;
							//*stackPtr_tmin++ = thit; *stackPtr_tmax++ = t_max;
							index = first_index;
							t_max = thit;

						}

					}
				}

				minDist[j] = distance;
			}

		}

	}

	//__global__ void octree_ray_gpu(Octree * d_oct) {
	//	printf("%d ", d_oct->text);
	//}

	//void make_cone(vec origin, vec dir, vector<vec>& output) {
	//	output.clear();
	//	//float halfAngle = 60;

	//	//int totalRings = 20;
	//	//int intervals = 5;	

	//	output.resize(RAYS_PER_ANGLE*ANGLE_INTERVALS + 1);

	//	//find perpendicular  vector as rotation axis

	//	vec rotateAxis = vec(1, 1, (-dir[0] - dir[1]) / dir[2]);
	//	if (dir[2] == 0) {
	//		rotateAxis = vec(0, 0, 1);
	//	}
	//	normalize(rotateAxis);
	//	normalize(dir);

	//	for (int j = 0; j < ANGLE_INTERVALS; j++) {
	//		vec ray1 = rotation(dir, CONE_ANGLE - (15 * j), rotateAxis);
	//		for (int i = 0; i < RAYS_PER_ANGLE; i++) {
	//			vec rayI = rotation(ray1, 360 * i / RAYS_PER_ANGLE, dir);
	//			output[(j*RAYS_PER_ANGLE) + i] = rayI;
	//		}
	//	}
	//	output[output.size() - 1] = dir;
	//}

	void make_cone_total(vec origin, vec dir, vector<vec>& output) {
		//output.clear();
		//float halfAngle = 60;

		//int totalRings = 20;
		//int intervals = 5;	
		//vector<vec> output;
		//output.resize(RAYS_PER_ANGLE*ANGLE_INTERVALS + 1);

		//find perpendicular  vector as rotation axis

		vec rotateAxis = vec(1, 1, (-dir[0] - dir[1]) / dir[2]);
		if (dir[2] == 0) {
			rotateAxis = vec(0, 0, 1);
		}
		normalize(rotateAxis);
		normalize(dir);

		for (int j = 0; j < ANGLE_INTERVALS; j++) {
			vec ray1 = rotation(dir, CONE_ANGLE - (15 * j), rotateAxis);
			for (int i = 0; i < RAYS_PER_ANGLE; i++) {
				vec rayI = rotation(ray1, 360 * i / RAYS_PER_ANGLE, dir);
				output.push_back(rayI);
			}
		}
		output.push_back(dir);
	}

	//Goal: Calculate the distance
	/*input:
	vectex:	vector start point
	dir:		vecter direction
	v1,v2,v3:	points of the face
	*/
	/*Moller��CTrumbore intersection algorithm
	p + t * d = (1-u-v) * p0 + u * p1 + v * p2
	t , u v are parameters
	p0 p1 p2 are apex of triangles
	p are point on ray, d are dir

	*/


	template < class T>
	float sdf_stat_mean(vector<T> input) {
		float sum = std::accumulate(input.begin(), input.end(), 0.0f);
		float mean = sum / input.size();
		float sq_sum = std::inner_product(input.begin(), input.end(), input.begin(), 0.0);
		float stdev = std::sqrt(sq_sum / input.size() - mean * mean);

		return mean;
	}

	template < class T>
	float sdf_stat_mean(vector<T> input, float median, float stdev) {
		vector<T> new_input;
		int n_stdev = 1;
		float lower_threshold = median - stdev*n_stdev;
		float upper_threshold = median + stdev*n_stdev;
		for (int i = 0; i < input.size(); i++) {
			if (input[i] >= lower_threshold && input[i] <= upper_threshold) {
				new_input.push_back(input[i]);
			}
		}

		float sum = std::accumulate(new_input.begin(), new_input.end(), 0.0f);
		float mean = sum / new_input.size();
		return mean;
	}

	template < class T>
	float sdf_stat_stdev(vector<T> input) {
		float sum = std::accumulate(input.begin(), input.end(), 0.0f);
		float mean = sum / input.size();
		float sq_sum = std::inner_product(input.begin(), input.end(), input.begin(), 0.0);
		float stdev = std::sqrt(sq_sum / input.size() - mean * mean);

		return stdev;
	}

	template < class T>
	float sdf_stat_median(vector<T> vals) {
		int n = vals.size();
		if (n & 1) {
			nth_element(vals.begin(),
				vals.begin() + n / 2,
				vals.end());
			return vals[n / 2];
		}
		else {
			nth_element(vals.begin(),
				vals.begin() + n / 2 - 1,
				vals.end());
			float tmp = vals[n / 2 - 1];
			nth_element(vals.begin(),
				vals.begin() + n / 2,
				vals.end());
			return 0.5f * (tmp + vals[n / 2]);
		}
	}

	//---------function for sdf-------------------------


	void TriMesh::need_sdf_gpu_1D() {
		std::clock_t begin = clock();
		std::cout << std::endl << "computing the SDF using gpu without loop: ";
		if (sdf_brute_gpu.size() == vertices.size())
			return;
		sdf_brute_gpu.resize(vertices.size());



		typedef long long llong;
		int nf = faces.size();
		int nv = vertices.size();
		int n_rays = 81;
		llong total_N = (llong)nf*(llong)nv *(llong)n_rays;
		std::cout << total_N << std::endl;
		llong nv_per_grid = ((llong)LONG_MAX) / (llong)(nf*n_rays);
		llong nf_per_grid = ((llong)LONG_MAX) / (llong)(nv*n_rays);
		if (nf>nf_per_grid) {
			std::cout << "greater than max block size" << endl;
		}
		else {
			nf_per_grid = nf;
		}
		if (nv>nv_per_grid) {
			std::cout << "greater than max block size" << endl;
		}
		else {
			nv_per_grid = nv;
		}


		//std::cout << "the max f " << nf_per_grid <<endl;
		//long n_blocks = LONG_MAX - 1;
		//	nf_per_grid = 2000;
		long n_grids = nf / nf_per_grid;

		long n_grids_v = nv / nv_per_grid;
		//std::cout << "the number of grids " << n_grids << endl;
		//	std::cout << n_grids << std::endl;


		need_faceNormals();

		//---allocate the points of faces and face normals on Memory
		float * p_faces_1pt[3], *p_faces_2pt[3], *p_faces_3pt[3], *p_face_normal[3];
		for (int j = 0; j < 3; j++) {
			p_faces_1pt[j] = new float[nf];
			p_faces_2pt[j] = new float[nf];
			p_faces_3pt[j] = new float[nf];
			p_face_normal[j] = new float[nf];

		}
		for (int j = 0; j < 3; j++) {
			for (int i = 0; i < nf; i++) {
				p_faces_1pt[j][i] = vertices[faces[i][0]][j];
				p_faces_2pt[j][i] = vertices[faces[i][1]][j];
				p_faces_3pt[j][i] = vertices[faces[i][2]][j];
				p_face_normal[j][i] = faceNormals[i][j];
			}
		}

		float * d_pt1[3], *d_pt2[3], *d_pt3[3], *d_face_normals[3];


		//allocate the points of faces and face normals on CUDA
		for (int i = 0; i < 3; i++) {
			hipMalloc((void**)&(d_pt1[i]), nf * sizeof(float));
			hipMemcpy(d_pt1[i], p_faces_1pt[i], nf * sizeof(float), hipMemcpyHostToDevice);

			hipMalloc((void**)&(d_pt2[i]), nf * sizeof(float));
			hipMemcpy(d_pt2[i], p_faces_2pt[i], nf * sizeof(float), hipMemcpyHostToDevice);

			hipMalloc((void**)&(d_pt3[i]), nf * sizeof(float));
			hipMemcpy(d_pt3[i], p_faces_3pt[i], nf * sizeof(float), hipMemcpyHostToDevice);

			hipMalloc((void**)&(d_face_normals[i]), nf * sizeof(float));
			hipMemcpy(d_face_normals[i], p_face_normal[i], nf * sizeof(float), hipMemcpyHostToDevice);
		}



		//make the total cones(nv * 81 )
		//	std::vector<vec> rays;
		std::vector<vec> vertices_rays;
		for (int i = 0; i < nv; i++) {
			vec inwardNormal = inwardNormals[i];
			vec vertex = vertices[i];
			make_cone_total(vertex, inwardNormal, vertices_rays);

		}
		int total_nRay = nv* n_rays;
		float * total_rayX = new float[total_nRay], *total_rayY = new float[total_nRay], *total_rayZ = new float[total_nRay];
		float *host_result = new float[total_nRay];
		//initialize the HOST pointers of total rays of vertices.
		for (int i = 0; i < total_nRay; i++) {
			total_rayX[i] = vertices_rays[i][0];
			total_rayY[i] = vertices_rays[i][1];
			total_rayZ[i] = vertices_rays[i][2];
			host_result[i] = 0;
		}
		//initialize the HOST pointers of vertices.
		float *host_v_x = new float[nv], *host_v_y = new float[nv], *host_v_z = new float[nv];
		float *host_normal_x = new float[nv], *host_normal_y = new float[nv], *host_normal_z = new float[nv];
		for (int i = 0; i < nv; i++) {
			host_v_x[i] = vertices[i][0];
			host_v_y[i] = vertices[i][1];
			host_v_z[i] = vertices[i][2];
			host_normal_x[i] = normals[i][0];
			host_normal_y[i] = normals[i][1];
			host_normal_z[i] = normals[i][2];
		}
		float * d_total_rayX, *d_total_rayY, *d_total_rayZ;
		float *d_v_x, *d_v_y, *d_v_z;
		float *d_normal_x, *d_normal_y, *d_normal_z;
		float * d_result;

		//locate and copy total rays
		gpuErrchk(hipMalloc((void**)&(d_total_rayX), total_nRay * sizeof(float)));
		gpuErrchk(hipMalloc((void**)&(d_total_rayY), total_nRay * sizeof(float)));
		gpuErrchk(hipMalloc((void**)&(d_total_rayZ), total_nRay * sizeof(float)));
		gpuErrchk(hipMemcpy(d_total_rayX, total_rayX, total_nRay * sizeof(float), hipMemcpyHostToDevice));
		gpuErrchk(hipMemcpy(d_total_rayY, total_rayY, total_nRay * sizeof(float), hipMemcpyHostToDevice));
		gpuErrchk(hipMemcpy(d_total_rayZ, total_rayZ, total_nRay * sizeof(float), hipMemcpyHostToDevice));

		//locate and copy vertices
		gpuErrchk(hipMalloc((void**)&(d_v_x), nv * sizeof(float)));
		gpuErrchk(hipMalloc((void**)&(d_v_y), nv * sizeof(float)));
		gpuErrchk(hipMalloc((void**)&(d_v_z), nv * sizeof(float)));
		gpuErrchk(hipMemcpy(d_v_x, host_v_x, nv * sizeof(float), hipMemcpyHostToDevice));
		gpuErrchk(hipMemcpy(d_v_y, host_v_y, nv * sizeof(float), hipMemcpyHostToDevice));
		gpuErrchk(hipMemcpy(d_v_z, host_v_z, nv * sizeof(float), hipMemcpyHostToDevice));

		//locate and copy normals
		gpuErrchk(hipMalloc((void**)&(d_normal_x), nv * sizeof(float)));
		gpuErrchk(hipMalloc((void**)&(d_normal_y), nv * sizeof(float)));
		gpuErrchk(hipMalloc((void**)&(d_normal_z), nv * sizeof(float)));
		gpuErrchk(hipMemcpy(d_normal_x, host_normal_x, nv * sizeof(float), hipMemcpyHostToDevice));
		gpuErrchk(hipMemcpy(d_normal_y, host_normal_y, nv * sizeof(float), hipMemcpyHostToDevice));
		gpuErrchk(hipMemcpy(d_normal_z, host_normal_z, nv * sizeof(float), hipMemcpyHostToDevice));

		//locate result
		gpuErrchk(hipMalloc((void**)&(d_result), total_nRay * sizeof(float)));
		gpuErrchk(hipMemcpy(d_result, host_result, total_nRay * sizeof(float), hipMemcpyHostToDevice));

		for (int i = 0; i <= (int)n_grids_v; i++) {
			//-----
			//initial the block size and block numbers.
			int blockSizeX = 128;
			int blockNumX = (nf + blockSizeX - 1) / blockSizeX;
			//blockNumX = nf / blockSizeX;

			int blockSizeY = 8;
			int blockNumY = (nv_per_grid + blockSizeY - 1) / blockSizeY;


			int blockSizeZ = 1;
			int blockNumZ = (n_rays + blockSizeZ - 1) / blockSizeZ;

			dim3 thread(blockSizeX, blockSizeY, blockSizeZ);
			dim3 blockNum(blockNumX, blockNumY, blockNumZ);
			//int i = 0;
			int nf_min = i*nv_per_grid;
			int nf_max = ((i + 1)*nv_per_grid) - 1;
			intersect_gpu_greater << <blockNum, thread >> > (d_pt1[0], d_pt1[1], d_pt1[2],
				d_pt2[0], d_pt2[1], d_pt2[2],
				d_pt3[0], d_pt3[1], d_pt3[2],
				d_face_normals[0], d_face_normals[1], d_face_normals[2],
				d_v_x, d_v_y, d_v_z,
				d_normal_x, d_normal_y, d_normal_z,
				d_total_rayX, d_total_rayY, d_total_rayZ,
				d_result,
				nv, n_rays, nf_max, nf_min);
			//if (i % (n_grids / 10) == 0)
			//	std::cout << "Finish " << i / (n_grids / 10) << "0% of ht SDF" << std::endl;
			//	gpuErrchk(hipMemcpy(host_result, d_result, total_nRay * sizeof(float), hipMemcpyDeviceToHost));
			//gpuErrchk(hipMemcpy(d_result, host_result, total_nRay * sizeof(float), hipMemcpyHostToDevice));
		}


		gpuErrchk(hipMemcpy(host_result, d_result, total_nRay * sizeof(float), hipMemcpyDeviceToHost));

		for (int i = 0; i < nv; i++) {
			vector<float>result;
			for (int j = 0; j < n_rays; j++) {
				if (host_result[i*n_rays + j] != 0)
					result.push_back(host_result[i*n_rays + j]);
			}
			if (result.size() == 0) {
				sdf_brute_gpu[i] = 0;// sdf_stat_mean(result);
			}
			else {
				sdf_brute_gpu[i] = sdf_stat_mean(result);
			}

		}

		int size0_dist_count = 0;


		//cuda free memory
		hipFree(d_normal_x); hipFree(d_normal_y); hipFree(d_normal_z);
		hipFree(d_v_x); hipFree(d_v_y); hipFree(d_v_z);
		hipFree(d_total_rayX); hipFree(d_total_rayY); hipFree(d_total_rayZ);
		hipFree(d_result);
		for (int i = 0; i < 3; i++) {
			hipFree(d_pt1[i]);
			hipFree(d_pt2[i]);
			hipFree(d_pt3[i]);
			hipFree(d_face_normals[i]);
		}
		std::cout << double(clock() - begin) / CLOCKS_PER_SEC << " s" << std::endl;
	}

	// use cuda to accelaterated sdf calculation.

	void TriMesh::need_sdf_kd_tree_gpu() {
		if (sdf.size() == vertices.size())
			return;
		sdf.resize(vertices.size());

		clock_t begin = clock();
		std::cout << std::endl << "computing the shape diameters using KD TREE with GPU: ";
		need_faceNormals();
		KD_tree kd_tree_pointer;
		trimesh::buildKDTree(kd_tree_pointer, faces, vertices, faceNormals);
		KD_tree_array * kd_array = KDTreeToArray(kd_tree_pointer);

		sdf_brute_gpu.resize(vertices.size());


		int n_tris = kd_array->pt1X_1d.size();
		typedef long long llong;
		int N_kd_array = kd_array->pt1X.size();
		int nv = vertices.size();
		int n_rays = 81;
		llong total_N = (llong)nv *(llong)n_rays;
		std::cout << total_N << std::endl;
		llong nv_per_grid = ((llong)LONG_MAX) / (llong)(n_rays);


		if (nv>nv_per_grid) {
			std::cout << "greater than max block size" << endl;
		}
		else {
			nv_per_grid = nv;
		}


		//std::cout << "the max f " << nf_per_grid <<endl;
		//long n_blocks = LONG_MAX - 1;
		//	nf_per_grid = 2000;


		long n_grids_v = nv / nv_per_grid;
		//std::cout << "the number of grids " << n_grids << endl;
		//	std::cout << n_grids << std::endl;





		float * d_pt1[3], *d_pt2[3], *d_pt3[3], *d_face_normals[3];


		//make the total cones(nv * 81 )
		//	std::vector<vec> rays;
		std::vector<vec> vertices_rays;
		for (int i = 0; i < nv; i++) {
			vec inwardNormal = inwardNormals[i];
			vec vertex = vertices[i];
			make_cone_total(vertex, inwardNormal, vertices_rays);

		}
		int total_nRay = nv* n_rays;
		float * total_rayX = new float[total_nRay], *total_rayY = new float[total_nRay], *total_rayZ = new float[total_nRay];
		float *host_result = new float[total_nRay];
		//initialize the HOST pointers of total rays of vertices.
		for (int i = 0; i < total_nRay; i++) {
			total_rayX[i] = vertices_rays[i][0];
			total_rayY[i] = vertices_rays[i][1];
			total_rayZ[i] = vertices_rays[i][2];
			host_result[i] = 0;
		}
		//initialize the HOST pointers of vertices.
		float *host_v_x = new float[nv], *host_v_y = new float[nv], *host_v_z = new float[nv];
		float *host_normal_x = new float[nv], *host_normal_y = new float[nv], *host_normal_z = new float[nv];
		for (int i = 0; i < nv; i++) {
			host_v_x[i] = vertices[i][0];
			host_v_y[i] = vertices[i][1];
			host_v_z[i] = vertices[i][2];
			host_normal_x[i] = normals[i][0];
			host_normal_y[i] = normals[i][1];
			host_normal_z[i] = normals[i][2];
		}
		float * d_total_rayX, *d_total_rayY, *d_total_rayZ;
		float *d_v_x, *d_v_y, *d_v_z;
		float *d_normal_x, *d_normal_y, *d_normal_z;
		float * d_result;

		//locate and copy total rays
		gpuErrchk(hipMalloc((void**)&(d_total_rayX), total_nRay * sizeof(float)));
		gpuErrchk(hipMalloc((void**)&(d_total_rayY), total_nRay * sizeof(float)));
		gpuErrchk(hipMalloc((void**)&(d_total_rayZ), total_nRay * sizeof(float)));
		gpuErrchk(hipMemcpy(d_total_rayX, total_rayX, total_nRay * sizeof(float), hipMemcpyHostToDevice));
		gpuErrchk(hipMemcpy(d_total_rayY, total_rayY, total_nRay * sizeof(float), hipMemcpyHostToDevice));
		gpuErrchk(hipMemcpy(d_total_rayZ, total_rayZ, total_nRay * sizeof(float), hipMemcpyHostToDevice));

		//locate and copy vertices
		gpuErrchk(hipMalloc((void**)&(d_v_x), nv * sizeof(float)));
		gpuErrchk(hipMalloc((void**)&(d_v_y), nv * sizeof(float)));
		gpuErrchk(hipMalloc((void**)&(d_v_z), nv * sizeof(float)));
		gpuErrchk(hipMemcpy(d_v_x, host_v_x, nv * sizeof(float), hipMemcpyHostToDevice));
		gpuErrchk(hipMemcpy(d_v_y, host_v_y, nv * sizeof(float), hipMemcpyHostToDevice));
		gpuErrchk(hipMemcpy(d_v_z, host_v_z, nv * sizeof(float), hipMemcpyHostToDevice));

		//locate and copy normals
		gpuErrchk(hipMalloc((void**)&(d_normal_x), nv * sizeof(float)));
		gpuErrchk(hipMalloc((void**)&(d_normal_y), nv * sizeof(float)));
		gpuErrchk(hipMalloc((void**)&(d_normal_z), nv * sizeof(float)));
		gpuErrchk(hipMemcpy(d_normal_x, host_normal_x, nv * sizeof(float), hipMemcpyHostToDevice));
		gpuErrchk(hipMemcpy(d_normal_y, host_normal_y, nv * sizeof(float), hipMemcpyHostToDevice));
		gpuErrchk(hipMemcpy(d_normal_z, host_normal_z, nv * sizeof(float), hipMemcpyHostToDevice));

		//locate result
		gpuErrchk(hipMalloc((void**)&(d_result), total_nRay * sizeof(float)));
		gpuErrchk(hipMemcpy(d_result, host_result, total_nRay * sizeof(float), hipMemcpyHostToDevice));

		//------------------------
		//--Locate the attributes in KD tree.
		float * d_p1x, *d_p1y, *d_p1z;
		float * d_p2x, *d_p2y, *d_p2z;
		float * d_p3x, *d_p3y, *d_p3z;
		float * d_fNx, *d_fNy, *d_fNz;



		gpuErrchk(hipMalloc((void**)&(d_p1x), n_tris * sizeof(float)));
		gpuErrchk(hipMalloc((void**)&(d_p1y), n_tris * sizeof(float)));
		gpuErrchk(hipMalloc((void**)&(d_p1z), n_tris * sizeof(float)));
		gpuErrchk(hipMalloc((void**)&(d_p2x), n_tris * sizeof(float)));
		gpuErrchk(hipMalloc((void**)&(d_p2y), n_tris * sizeof(float)));
		gpuErrchk(hipMalloc((void**)&(d_p2z), n_tris * sizeof(float)));
		gpuErrchk(hipMalloc((void**)&(d_p3x), n_tris * sizeof(float)));
		gpuErrchk(hipMalloc((void**)&(d_p3y), n_tris * sizeof(float)));
		gpuErrchk(hipMalloc((void**)&(d_p3z), n_tris * sizeof(float)));
		gpuErrchk(hipMalloc((void**)&(d_fNx), n_tris * sizeof(float)));
		gpuErrchk(hipMalloc((void**)&(d_fNy), n_tris * sizeof(float)));
		gpuErrchk(hipMalloc((void**)&(d_fNz), n_tris * sizeof(float)));

		gpuErrchk(hipMemcpy(d_p1x, &(kd_array->pt1X_1d[0]), n_tris * sizeof(float), hipMemcpyHostToDevice));
		gpuErrchk(hipMemcpy(d_p1y, &(kd_array->pt1Y_1d[0]), n_tris * sizeof(float), hipMemcpyHostToDevice));
		gpuErrchk(hipMemcpy(d_p1z, &(kd_array->pt1Z_1d[0]), n_tris * sizeof(float), hipMemcpyHostToDevice));

		gpuErrchk(hipMemcpy(d_p2x, &(kd_array->pt2X_1d[0]), n_tris * sizeof(float), hipMemcpyHostToDevice));
		gpuErrchk(hipMemcpy(d_p2y, &(kd_array->pt2Y_1d[0]), n_tris * sizeof(float), hipMemcpyHostToDevice));
		gpuErrchk(hipMemcpy(d_p2z, &(kd_array->pt2Z_1d[0]), n_tris * sizeof(float), hipMemcpyHostToDevice));

		gpuErrchk(hipMemcpy(d_p3x, &(kd_array->pt3X_1d[0]), n_tris * sizeof(float), hipMemcpyHostToDevice));
		gpuErrchk(hipMemcpy(d_p3y, &(kd_array->pt3Y_1d[0]), n_tris * sizeof(float), hipMemcpyHostToDevice));
		gpuErrchk(hipMemcpy(d_p3z, &(kd_array->pt3Z_1d[0]), n_tris * sizeof(float), hipMemcpyHostToDevice));

		gpuErrchk(hipMemcpy(d_fNx, &(kd_array->fNomarlX_1d[0]), n_tris * sizeof(float), hipMemcpyHostToDevice));
		gpuErrchk(hipMemcpy(d_fNy, &(kd_array->fNomarlY_1d[0]), n_tris * sizeof(float), hipMemcpyHostToDevice));
		gpuErrchk(hipMemcpy(d_fNz, &(kd_array->fNomarlZ_1d[0]), n_tris * sizeof(float), hipMemcpyHostToDevice));


		float * d_split;
		int * d_triCount, *d_split_axis;
		int * d_tri_index;
		gpuErrchk(hipMalloc((void**)  &(d_split), sizeof(float) *N_kd_array));
		gpuErrchk(hipMalloc((void**)  &(d_triCount), sizeof(int) * N_kd_array));
		gpuErrchk(hipMalloc((void**)  &(d_split_axis), sizeof(int) * N_kd_array));
		gpuErrchk(hipMalloc((void**)  &(d_tri_index), sizeof(int) * N_kd_array));

		gpuErrchk(hipMemcpy(d_split, &(kd_array->split[0]), sizeof(float) * N_kd_array, hipMemcpyHostToDevice));
		gpuErrchk(hipMemcpy(d_triCount, &(kd_array->triCount[0]), sizeof(int)* N_kd_array, hipMemcpyHostToDevice));
		gpuErrchk(hipMemcpy(d_split_axis, &(kd_array->split_axis[0]), sizeof(int)* N_kd_array, hipMemcpyHostToDevice));
		gpuErrchk(hipMemcpy(d_tri_index, &(kd_array->triIndex[0]), sizeof(int)* N_kd_array, hipMemcpyHostToDevice));

		float *d_max, *d_min;
		gpuErrchk(hipMalloc((void**)  &(d_max), sizeof(float) * 3));
		gpuErrchk(hipMalloc((void**)  &(d_min), sizeof(float) * 3));
		gpuErrchk(hipMemcpy(d_max, &(kd_array->max), sizeof(float) * 3, hipMemcpyHostToDevice));
		gpuErrchk(hipMemcpy(d_min, &(kd_array->min), sizeof(float) * 3, hipMemcpyHostToDevice));

		int blockSizeX = 128;
		int blockNumX = (nv + blockSizeX - 1) / blockSizeX;
		//blockNumX = nf / blockSizeX;

		int blockSizeY = 8;
		int blockNumY = (n_rays + blockSizeY - 1) / blockSizeY;

		dim3 thread(blockSizeX, blockSizeY);
		dim3 blockNum(blockNumX, blockNumY);
		std::cout << double(clock() - begin) / CLOCKS_PER_SEC << " s" << std::endl;
		intersect_gpu_kdtree << <blockNum, thread >> > (d_p1x, d_p1y, d_p1z,
			d_p2x, d_p2y, d_p2z,
			d_p3x, d_p3y, d_p3z,
			d_fNx, d_fNy, d_fNz,
			d_split, d_split_axis, d_triCount, d_tri_index,
			d_max, d_min,
			d_v_x, d_v_y, d_v_z,
			d_normal_x, d_normal_y, d_normal_z,
			d_total_rayX, d_total_rayY, d_total_rayZ,
			d_result,
			nv, n_rays, N_kd_array

			);

		gpuErrchk(hipMemcpy(host_result, d_result, total_nRay * sizeof(float), hipMemcpyDeviceToHost));

		for (int i = 0; i < nv; i++) {
			vector<float>result;
			for (int j = 0; j < n_rays; j++) {
				if (host_result[i*n_rays + j] >= 0)
					result.push_back(host_result[i*n_rays + j]);
				else {
					result.push_back(bsphere.r * 2);
				}

			}
			if (result.size() == 0) {
				sdf[i] = 0;// sdf_stat_mean(result);
			}
			else {
				sdf[i] = sdf_stat_mean(result);
			}

		}
		std::cout << double(clock() - begin) / CLOCKS_PER_SEC << " s" << std::endl;
	}


}//end of namespace

 